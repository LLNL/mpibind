/******************************************************
 * Edgar A. Leon
 * Lawrence Livermore National Laboratory
 ******************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#ifdef HAVE_AMD_GPUS
#include "hip/hip_runtime.h"
#endif 

#define MAX_PCI_LEN 20

int main(int argc, char *argv[])
{
    char str[] = "1,7"; 

    // Copy VISDEVS string since strtok modifies the input string 
    char tmp[strlen(str)]; 
    strcpy(tmp, str);

    /* Get list size */ 
    int i, idevs = 0; 
    char *token = strtok(tmp, ",");
    while( token != NULL ) {
        idevs++; 
        token = strtok(NULL, ",");
    }

    /* Convert VISDEVS list into ints */ 
    //int i=0, visdevs[idevs];
    //strcpy(tmp, str);
    //token = strtok(tmp, ",");
    //while( token != NULL ) {
    //    visdevs[i++] = atoi(token);
    //    token = strtok(NULL, ",");
    //}

    // Don't invoke any GPU calls before resetting the environment!
    // Otherwise, there's no effect of setting VISIBLE_DEVICES. 
    //cudaGetDeviceCount(&ndevs);
    //printf("Initial num. devices %d\n", ndevs); 

    printf("Resetting environment to devices %s\n", str); 
    unsetenv("ROCR_VISIBLE_DEVICES");
    unsetenv("HIP_VISIBLE_DEVICES");
    unsetenv("CUDA_VISIBLE_DEVICES");
#ifdef HAVE_AMD_GPUS
    setenv("ROCR_VISIBLE_DEVICES", str, 1);
#else 
    setenv("CUDA_VISIBLE_DEVICES", str, 1);
#endif

    int odevs=-1; 
    hipGetDeviceCount(&odevs);
    printf("Modified num. devices %d\n", odevs); 

    /* Get device PCI ID */ 
    char pci[MAX_PCI_LEN]; 
    for (i=0; i<idevs; i++) {
        pci[0] = '\0'; 
        hipDeviceGetPCIBusId(pci, MAX_PCI_LEN, i);
        printf("PCI ID of device %d = %s\n", i, pci);
    }

    return 0; 
}