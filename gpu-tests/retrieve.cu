/******************************************************
 * Edgar A. Leon
 * Lawrence Livermore National Laboratory
 ******************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#ifdef HAVE_AMD_GPUS
#include "hip/hip_runtime.h"
#endif 

#define MAX_PCI_LEN 20

void chooseDevPartial(int dev)
{
  int odev=-1;
  int busId=-1, deviceId=-1, domainId=-1;
  char pci[MAX_PCI_LEN];
  hipDeviceProp_t prop; 

  // Get selected device properties 
  hipDeviceGetPCIBusId(pci, MAX_PCI_LEN, dev);
  sscanf(pci, "%04x:%02x:%02x", &domainId, &busId, &deviceId);
  
   // Partially fill device properties and match 
   memset(&prop, 0, sizeof(hipDeviceProp_t));
   prop.pciDomainID = domainId; 
   prop.pciBusID = busId;
   prop.pciDeviceID = deviceId; 
   
   hipChooseDevice(&odev, &prop);
   printf("Partial match of device %d: device %d\n", dev, odev);
   printf("\tInput: DomainID=0x%x BusId=0x%x DeviceId=0x%x\n",
    domainId, busId, deviceId); 
   if (dev != odev)
     printf("\tError: ChooseDevice did not match the correct device\n");
}

void chooseDevFull(int dev)
{
  int odev=-1;
  hipDeviceProp_t prop; 

  // Get all device properties 
  hipGetDeviceProperties(&prop, dev); 

  hipChooseDevice(&odev, &prop);
  printf("Full match of device %d: device %d\n", dev, odev);
  printf("\tInput: DomainID=0x%x BusId=0x%x DeviceId=0x%x\n",
	 prop.pciDomainID, prop.pciBusID, prop.pciDeviceID); 
#ifndef HAVE_AMD_GPUS
  // HIP does not have a uuid field! 
  printf("\t       UUID=0x%x\n", prop.uuid); 
#endif

  if (dev != odev)
    printf("\tError: ChooseDevice did not match the correct device\n");
}

void getDevByPCI(int dev, char *pci)
{
  int pciBusID=-1, pciDeviceID=-1, pciDomainID=-1;
  int odev=-1; 
    
  sscanf(pci, "%04x:%02x:%02x", &pciDomainID, &pciBusID, &pciDeviceID);

  // PCI ID: String in one of the following forms: 
  // [domain]:[bus]:[device].[function] 
  // [domain]:[bus]:[device] 
  // [bus]:[device].[function] 
  // where domain, bus, device, and function are all hex values
  hipDeviceGetByPCIBusId(&odev, pci);

  printf("GetbyPCI match of device %d: device %d\n", dev, odev);
  printf("\tInput: DomainID=0x%x BusId=0x%x DeviceId=0x%x\n",
    pciDomainID, pciBusID, pciDeviceID); 
  if (odev != dev)
    printf("Error: GetByPCI did not match the correct device\n");
}


int main(int argc, char *argv[])
{
  int dev, ndevs; 
  char pci[MAX_PCI_LEN];

  
  hipGetDeviceCount(&ndevs);
  if (ndevs <= 0) {
    printf("No devices found\n"); 
    return 0; 
  }

  // Select input device
  // Avoid choosing device 0, if possible, to enhance testing
  // dev = 1; 
  dev = ndevs-1; 

  hipDeviceGetPCIBusId(pci, MAX_PCI_LEN, dev);
  printf("PCI ID of device %d = %s\n", dev, pci);

  getDevByPCI(dev, pci);

  chooseDevPartial(dev);

  chooseDevFull(dev);

  hipSetDevice(dev);
    
  return 0; 
}

