/******************************************************
 * Edgar A. Leon
 * Lawrence Livermore National Laboratory
 ******************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hwloc.h>

#include <sys/wait.h>
#include <unistd.h>

#ifdef HAVE_AMD_GPUS
#include "hip/hip_runtime.h"
#endif 

#define MAX_PCI_LEN 20
#define MAX_STR_LEN 512


int obj_attr_snprintf(char *str, size_t size, hwloc_obj_t obj, 
                      int verbose)
{
  int nc=0; 

  if (obj->type == HWLOC_OBJ_OS_DEVICE)
  switch (obj->attr->osdev.type) {
    case HWLOC_OBJ_OSDEV_COPROC :
      nc += hwloc_obj_type_snprintf(str+nc, size-nc, obj, 1);
      nc += snprintf(str+nc, size-nc, ": name=%s ", obj->name);
      nc += snprintf(str+nc, size-nc, "subtype=%s ", obj->subtype);
      nc += snprintf(str+nc, size-nc, "GPUModel=%s ", 
        hwloc_obj_get_info_by_name(obj, "GPUModel"));
      nc += snprintf(str+nc, size-nc, "   ");
      /* Get obj->infos in one shot */ 
      nc += hwloc_obj_attr_snprintf(str+nc, size-nc, obj, " ", verbose);
      break; 

    default: 
      break;
  }

  
  return nc;
}
 


void set_vis_devs(char *str)
{
  // Don't invoke any GPU calls before resetting the environment!
  // Otherwise, there's no effect of setting VISIBLE_DEVICES. 
  //cudaGetDeviceCount(&ndevs);
  //printf("Initial num. devices %d\n", ndevs); 

  printf("Resetting environment to devices %s\n", str); 
  unsetenv("ROCR_VISIBLE_DEVICES");
  unsetenv("HIP_VISIBLE_DEVICES");
  unsetenv("CUDA_VISIBLE_DEVICES");
#ifdef HAVE_AMD_GPUS
    setenv("ROCR_VISIBLE_DEVICES", str, 1);
#else 
    setenv("CUDA_VISIBLE_DEVICES", str, 1);
#endif
}


void print_devices(hwloc_topology_t topo)
{
  char str[MAX_STR_LEN]; 
  hwloc_obj_t obj = NULL;
  while ( (obj=hwloc_get_next_obj_by_type(topo, HWLOC_OBJ_OS_DEVICE, obj)) != NULL ) 
    if (obj->attr->osdev.type == HWLOC_OBJ_OSDEV_COPROC) {
      str[0] = '\0'; 
      obj_attr_snprintf(str, MAX_STR_LEN, obj, 0);
      printf("%s\n", str);
  }
}

int get_list_len(char *lst)
{
  // Copy VISDEVS string since strtok modifies the input string 
  char tmp[strlen(lst)]; 
  strcpy(tmp, lst);

  /* Get list size */ 
  int idevs = 0; 
  char *token = strtok(tmp, ",");
  while( token != NULL ) {
    idevs++; 
    token = strtok(NULL, ",");
  }

  return idevs; 
}


void test_wdup(char *visdevs, hwloc_topology_t topo)
{
  set_vis_devs(visdevs); 

  hwloc_topology_t topo2; 
  printf("Duplicating the topology\n"); 
  hwloc_topology_dup(&topo2, topo); 

  set_vis_devs(visdevs); 

  print_devices(topo2);
  hwloc_topology_destroy(topo2);    
}

void test_wfork(char *vds)
{
  set_vis_devs(vds); 
  pid_t cpid = fork();
  
  if (cpid == 0) {
    unsetenv("ROCR_VISIBLE_DEVICES");
    unsetenv("HIP_VISIBLE_DEVICES");
    printf("Child:\n");
    set_vis_devs(vds); 

    hwloc_topology_t topo;
    hwloc_topology_init(&topo);
    hwloc_topology_set_io_types_filter(topo,
        HWLOC_TYPE_FILTER_KEEP_IMPORTANT);
    hwloc_topology_load(topo);
    print_devices(topo);
    hwloc_topology_destroy(topo);    

    exit(0);
  } else if (cpid > 0) {
    printf("Parent: Nothing to do but wait...\n");
    wait(NULL);
  } else {
    printf("fork() failed\n");
  }
}

void test_wnew_topo(char *vds)
{
  set_vis_devs(vds); 

  hwloc_topology_t topo;
  hwloc_topology_init(&topo);
  hwloc_topology_set_io_types_filter(topo,
      HWLOC_TYPE_FILTER_KEEP_IMPORTANT);
  hwloc_topology_load(topo);
  print_devices(topo);
  hwloc_topology_destroy(topo);    
}


void test_wdev_api(char *vds)
{
  int i, odevs=-1; 
  /* Cannot call the device driver before settting 
     VISIBLE DEVICES. Otherwise, the devices are set
     and cannot be changed */ 
  //cudaGetDeviceCount(&odevs);
  //printf("Modified num. devices %d\n", odevs); 

  set_vis_devs(vds); 
  hipGetDeviceCount(&odevs);
  printf("Modified num. devices %d\n", odevs); 

  /* Get device PCI ID */ 
  char pci[MAX_PCI_LEN]; 
  for (i=0; i<odevs; i++) {
      pci[0] = '\0'; 
      hipDeviceGetPCIBusId(pci, MAX_PCI_LEN, i);
      printf("PCI ID of device %d = %s\n", i, pci);
  }
}

void test_wfork_api(char *vds)
{
  int i, odevs=-1; 
  /* Don't call into device functions until 
     after setting visible devices */ 
  //cudaGetDeviceCount(&odevs);
  //printf("Num. devices %d\n", odevs); 

  set_vis_devs(vds); 
  hipGetDeviceCount(&odevs);
  printf("Num. devices %d\n", odevs); 

  pid_t cpid = fork();
  
  if (cpid == 0) {
    unsetenv("ROCR_VISIBLE_DEVICES");
    unsetenv("HIP_VISIBLE_DEVICES");
    printf("Child:\n");
    set_vis_devs(vds); 

    hipGetDeviceCount(&odevs);
    printf("Num. devices %d\n", odevs); 
    /* Get device PCI ID */ 
    char pci[MAX_PCI_LEN]; 
    for (i=0; i<odevs; i++) {
      pci[0] = '\0'; 
      hipDeviceGetPCIBusId(pci, MAX_PCI_LEN, i);
      printf("PCI ID of device %d = %s\n", i, pci);
    }
    
    exit(0);
  } else if (cpid > 0) {
    printf("Parent: Nothing to do but wait...\n");
    wait(NULL);
  } else {
    printf("fork() failed\n");
  }
}



/* Lessons learned: 
   1. Setting VISIBLE DEVICES in the context of hwloc: 
      The environmnet variables must be set before the 
      first time the topology is loaded. 
   2. Setting VISIBLE DEVICES in the context of device API calls: 
      The environment variables must be called before the 
      first invocation of a device function. 
   3. Using fork does not really allows to overwrite the points
      above. 
   4. hwloc loading a topology has the same effect as calling 
      a device function, i.e., after this setting VISIBLE 
      DEVICES is too late. 
 */ 

int main(int argc, char *argv[])
{
  char vds[] = "1"; 
  //int idevs = get_list_len(vds); 

  hwloc_topology_t topo;
  hwloc_topology_init(&topo);
  /* OS devices are filtered by default, enable to see GPUs */
  hwloc_topology_set_type_filter(topo, HWLOC_OBJ_OS_DEVICE,
      HWLOC_TYPE_FILTER_KEEP_IMPORTANT);
  /* Include PCI devices to determine whether two GPUs                          
     are the same device, i.e., opencl1d1 and cuda1 */
  hwloc_topology_set_type_filter(topo, HWLOC_OBJ_PCI_DEVICE,
       HWLOC_TYPE_FILTER_KEEP_IMPORTANT);

  /* Setting visible devices must be done before 
     loading the topology the first time! */ 
  set_vis_devs(vds); 

  /* If testing whether VISIBLE DEVICES work with 
     the device API functions, don't load the topology
     because this set the devices and can't be changed later */ 
  hwloc_topology_load(topo);
  //print_devices(topo);


#if 1
  test_wnew_topo(vds); 
#endif 
#if 0
  test_wdup(vds, topo);
#endif
#if 0
  test_wfork(vds);
#endif
#if 0
  test_wdev_api(vds);
#endif
#if 0
  test_wfork_api(vds);
#endif 

  hwloc_topology_destroy(topo);

  return 0; 
}