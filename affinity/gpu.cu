/***********************************************************
 * Edgar A. Leon
 * Lawrence Livermore National Laboratory 
 ***********************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>      /* Documentation in hip_runtime_api.h */ 
#include "affinity.h"          /* Do not perform name mangling */ 


int get_gpu_count()
{
  /* 
     Surprinsingly, I must set 'count' to zero before
     passing it to hipGetDeviceCount(&count)
     If CUDA_VISIBLE_DEVICES is set to '', calling 
     this function will not set a value for count. 
     Then, count will be used uninitialized and 
     most likely the program will segfault. 
  */ 
  int count=0;

  hipGetDeviceCount(&count);

  return count;
}


int get_gpu_pci_id(int dev)
{
  int value = -1; 
  hipError_t err = hipDeviceGetAttribute(&value, hipDeviceAttributePciBusId, dev);
  
  if ( err )
    fprintf(stderr, "Could not get PCI ID for GPU %d\n", dev);

  return value; 
}


int get_gpu_affinity(char *buf)
{
  int count=0;
  hipError_t err;
  hipDeviceProp_t prop;
  hipGetDeviceCount(&count);
  
  int nc=0; 
  int i, value; 
  for (i=0; i<count; i++) {
#if 1
    err = hipGetDeviceProperties(&prop, i);
    if ( err ) {
      fprintf(stderr, "Could not get info for GPU %d\n", i);
      return -1;
    }
    nc += sprintf(buf+nc, "%04x:%02x ", prop.pciDomainID, prop.pciBusID);
#else
    // [domain]:[bus]:[device].[function]
    char pcibusid[64];
    hipDeviceGetPCIBusId(pcibusid, 64, i);
    nc += sprintf(buf+nc, "%s", pcibusid);
#endif
  }
  nc += sprintf(buf+nc, "\n"); 
  
  return nc; 
}


int get_gpu_info(int devid, char *buf)
{
  hipDeviceProp_t prop;
  hipError_t err; 
  int nc = 0;
  
  err = hipGetDeviceProperties(&prop, devid);
  if ( err ) {
    fprintf(stderr, "Could not get info for GPU %d\n", devid);
    return -1;
  }

  float ghz = prop.clockRate / 1000.0 / 1000.0; 
#if 1
  nc += sprintf(buf+nc, "\tName: %s\n", prop.name);
  nc += sprintf(buf+nc, "\tPCI domain ID 0x%x\n", prop.pciDomainID);
  nc += sprintf(buf+nc, "\tPCI bus ID: 0x%x\n", prop.pciBusID);
  nc += sprintf(buf+nc, "\tPCI device ID 0x%x\n", prop.pciDeviceID);
  nc += sprintf(buf+nc, "\tMemory: %lu GB\n", prop.totalGlobalMem >> 30);
  nc += sprintf(buf+nc, "\tMultiprocessor count: %d\n", prop.multiProcessorCount);
  nc += sprintf(buf+nc, "\tClock rate: %.3f Ghz\n", ghz); 
  nc += sprintf(buf+nc, "\tCompute capability: %d.%d\n",
		prop.major, prop.minor);
  nc += sprintf(buf+nc, "\tECC enabled: %d\n", prop.ECCEnabled);
#else
  nc += sprintf(buf+nc, "\t0x%.2x: %s, %lu GB Mem, "
		"%d Multiprocessors, %.3f GHZ, %d.%d CC\n",
		prop.pciBusID, prop.name, prop.totalGlobalMem >> 30,
		prop.multiProcessorCount, ghz, prop.major, prop.minor); 
#endif
  
  return nc; 
}


int get_gpu_info_all(char *buf)
{
  hipError_t err; 
  int i, myid, count=0, value;
  int nc=0; 
  
  hipGetDeviceCount(&count);
  err = hipGetDevice(&myid);
  if ( err ) {
    fprintf(stderr, "Could not get default device\n");
    return -1; 
  }

  char pcibusid[SHORT_STR_SIZE];
  hipDeviceGetPCIBusId(pcibusid, sizeof(pcibusid), myid);
  nc += sprintf(buf+nc, "\tDefault device: %s\n", pcibusid);
  
  for (i=0; i<count; i++) {
    //nc += sprintf(buf+nc, "\t--\n"); 
    nc += get_gpu_info(i, buf+nc);
  }
  
  return nc; 
}




